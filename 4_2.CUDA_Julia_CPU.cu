#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    hipComplex( float a, float b ) : r(a), i(b)  {}
    float magnitude2( void ) { return r * r + i * i; }
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

int julia( int x, int y ) { 
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2); //픽셀 좌표를 복소수 공간으로 변경
    float jy = scale * (float)(DIM/2 - y)/(DIM/2); //픽셀 좌표를 복소수 공간으로 변경

    hipComplex c(-0.8, 0.156); //줄리아 공식에 사용되는 상수 c에 대한 임의이 값, 조정가능
    hipComplex a(jx, jy);

    // 줄리아 공식으로 판별 1000을 넘으면 0 안넘으면 1 반환
    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

void kernel( unsigned char *ptr ){
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;

            int juliaValue = julia( x, y );
            ptr[offset*4 + 0] = 255 * juliaValue;
            ptr[offset*4 + 1] = 0;
            ptr[offset*4 + 2] = 0;
            ptr[offset*4 + 3] = 255;
        }
    }
 }

int main( void ) {
    CPUBitmap bitmap( DIM, DIM );
    unsigned char *ptr = bitmap.get_ptr();

    kernel( ptr );

    bitmap.display_and_exit();
}
